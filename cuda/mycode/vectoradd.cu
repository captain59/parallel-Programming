#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void arradd(const int *md, const int *nd, const int *pd, int size){
	int myid = blockDim.x * blockIdx.x + threadIdx.x;
	p[myid] = md[myid] + nd[myid];
}
int main(){
	int size = 200*sizeof(int);
	int i = 0;
	int *m, *n, *p;
	//Allocating memory on CPU
	m = (int*)malloc(size);
	n = (int*)malloc(size);
	p = (int*)malloc(size);
	for(i=0; i<200; i++)
		m[i]=i, n[i]=i, p[i]=0;

	// Allocating memery on the Gpu
	int *md, *nd, *pd;
	hipMalloc(&md, size);
	// (destination, sources, n.o of bytes, direction)
	hipMemcpy(md, m, size, hipMemcpyHostToDevice);

	hipMalloc(&nd, size);
	hipMemcpy(nd, n, size, hipMemcpyHostToDevice);

	hipMalloc(&pd, size);
	// no need to allocate size as addition will generate which transfer from gpu to cpu
	hipMemcpy(pd, size);

	dim3 DimGrid(1, 1);
	dim3 DimBlock(200, 1);

	arradd<<<DimGrid, DimBlock>>(md, nd, pd, size);

	hipMemcpy(p, pd, size, hipMemcpyDeviceToHost);
	for(int i=0;i < 200; i++)
		printf("%d\n", p[i]);
	hipFree(md);
	hipFree(nd);
	hipFree(pd);
	free(m);
	free(n);
	free(p);
	// Reser the Device and exit
	hipError_t err = hipDeviceReset();
	if( err != hipSuccess){
		printf("Failed to deinitialize the device error %s \n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	printf("Done\n");
	return 0;
}